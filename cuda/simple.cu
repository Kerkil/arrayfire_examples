#include "hip/hip_runtime.h"
#include <stdio.h>
#include <arrayfire.h>
#include "simple.h"

const int num = 1024;

int main()
{
    // Generate input data
    af::array x = af::randu(num, 1);

    // Create empty array with required dimensions
    af::array y = af::array(num, 1);

    // Get device pointers
    float *d_x = x.device<float>();
    float *d_y = y.device<float>();

    // Finish the tasks arrayfire was doing
    af::sync();

    // Launch kernel to do the following operations
    // y = sin(x)^2 + cos(x)^2
    launch_simple_kernel(d_y, d_x, num);

    // Tell arrayfire it controls the pointers again
    x.unlock();
    y.unlock();

    // Check for errors
    // sin(x)^ + cos(x)^2 == 1
    // The following should print 0
    float err = af::sum<float>(y - 1);
    printf("Error: %f\n", err);

    return 0;
}
