#include "hip/hip_runtime.h"
#include <stdio.h>
#include <arrayfire.h>
#include "simple.h"

const int num = 1024;

int main()
{
    // Generate input data
    af::array x = af::randu(num, 1);

    // Get device pointers
    float *d_x = x.device<float>();

    // Allocate data needed for output
    float *d_y = af::array::alloc<float>(num);

    // Finish the tasks arrayfire was doing
    af::sync();

    // Launch kernel to do the following operations
    // y = sin(x)^2 + con(x)^2
    launch_simple_kernel(d_y, d_x, num);

    // Create arrays from output data
    af::array y(num, d_y);

    // Check for errors
    // sin(x)^ + cos(x)^2 == 1
    // The following should print 0
    float err = af::sum<float>(y - 1);
    printf("Error: %f\n", err);
    return 0;
}
